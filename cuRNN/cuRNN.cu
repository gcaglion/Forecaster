#include "hip/hip_runtime.h"
#include <cuRNN.h>

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
	if (stat != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
	}
}

#define cudnnErrCheck(stat) { cudnnErrCheck_((stat), __FILE__, __LINE__); }
void cudnnErrCheck_(hipdnnStatus_t stat, const char *file, int line) {
	if (stat != HIPDNN_STATUS_SUCCESS) {
		fprintf(stderr, "cuDNN Error: %s %s %d\n", hipdnnGetErrorString(stat), file, line);
	}
}

__global__ void initGPUData_ker(float *data, int numElements, float value) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < numElements) {
		data[tid] = value;
	}
}

void initGPUData(float *data, int numElements, float value) {
	dim3 gridDim;
	dim3 blockDim;

	blockDim.x = 1024;
	gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

	initGPUData_ker << < gridDim, blockDim >> > (data, numElements, value);
}

int setupGPU(
	int seqLength, int inputSize, int batchCount, int batchSize, int numLayers, int hiddenSize, bool bidirectional, int mode, float dropout,
	float** hSample, float** hTarget/*,
	void* x, void* hx, void* cx, void* y, void* hy, void* cy, void* dx, void* dhx, void* dcx, void* dy, void* dhy, void* dcy,
	hipdnnTensorDescriptor_t* xDesc, hipdnnTensorDescriptor_t* yDesc, hipdnnTensorDescriptor_t* dxDesc, hipdnnTensorDescriptor_t* dyDesc*/
	) {

	//=== 0. Create cudnn context
	hipdnnHandle_t cudnnHandle;
	cudnnErrCheck(hipdnnCreate(&cudnnHandle));
	//===

	//=== 1. Create Data structures on GPU
	void* x;
	void* hx = NULL;
	void* cx = NULL;

	void* y;
	void* hy = NULL;
	void* cy = NULL;

	void* dx;
	void* dhx = NULL;
	void* dcx = NULL;

	void* dy;
	void* dhy = NULL;
	void* dcy = NULL;

	// Memory allocation. hx, cx, dhx, dcx, hy, cy, dhy and dcy can be NULL.
	cudaErrCheck(hipMalloc((void**)&x, seqLength * inputSize * batchSize * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&hx, numLayers * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&cx, numLayers * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));

	cudaErrCheck(hipMalloc((void**)&dx, seqLength * inputSize * batchSize * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&dhx, numLayers * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&dcx, numLayers * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));

	cudaErrCheck(hipMalloc((void**)&y, seqLength * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&hy, numLayers * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&cy, numLayers * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));

	cudaErrCheck(hipMalloc((void**)&dy, seqLength * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&dhy, numLayers * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&dcy, numLayers * hiddenSize * batchSize * (bidirectional ? 2 : 1) * sizeof(float)));

	// Set up tensor descriptors. x/y/dx/dy are arrays, one per time step.
	hipdnnTensorDescriptor_t *xDesc, *yDesc, *dxDesc, *dyDesc;
	hipdnnTensorDescriptor_t hxDesc, cxDesc;
	hipdnnTensorDescriptor_t hyDesc, cyDesc;
	hipdnnTensorDescriptor_t dhxDesc, dcxDesc;
	hipdnnTensorDescriptor_t dhyDesc, dcyDesc;

	xDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
	yDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
	dxDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
	dyDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));

	// In this example dimA[1] is constant across the whole sequence
	// This isn't required, all that is required is that it does not increase.
	int dimA[3];
	int strideA[3];

	for (int i = 0; i < seqLength; i++) {
		cudnnErrCheck(hipdnnCreateTensorDescriptor(&xDesc[i]));
		cudnnErrCheck(hipdnnCreateTensorDescriptor(&yDesc[i]));
		cudnnErrCheck(hipdnnCreateTensorDescriptor(&dxDesc[i]));
		cudnnErrCheck(hipdnnCreateTensorDescriptor(&dyDesc[i]));

		dimA[0] = batchSize;
		dimA[1] = inputSize;
		dimA[2] = 1;

		strideA[0] = dimA[2] * dimA[1];
		strideA[1] = dimA[2];
		strideA[2] = 1;

		cudnnErrCheck(hipdnnSetTensorNdDescriptor(xDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
		cudnnErrCheck(hipdnnSetTensorNdDescriptor(dxDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));

		dimA[0] = batchSize;
		dimA[1] = bidirectional ? hiddenSize * 2 : hiddenSize;
		dimA[2] = 1;

		strideA[0] = dimA[2] * dimA[1];
		strideA[1] = dimA[2];
		strideA[2] = 1;

		cudnnErrCheck(hipdnnSetTensorNdDescriptor(yDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
		cudnnErrCheck(hipdnnSetTensorNdDescriptor(dyDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	}


	dimA[0] = numLayers * (bidirectional ? 2 : 1);
	dimA[1] = batchSize;
	dimA[2] = hiddenSize;

	strideA[0] = dimA[2] * dimA[1];
	strideA[1] = dimA[2];
	strideA[2] = 1;

	cudnnErrCheck(hipdnnCreateTensorDescriptor(&hxDesc));
	cudnnErrCheck(hipdnnCreateTensorDescriptor(&cxDesc));
	cudnnErrCheck(hipdnnCreateTensorDescriptor(&hyDesc));
	cudnnErrCheck(hipdnnCreateTensorDescriptor(&cyDesc));
	cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhxDesc));
	cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcxDesc));
	cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhyDesc));
	cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcyDesc));

	cudnnErrCheck(hipdnnSetTensorNdDescriptor(hxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	cudnnErrCheck(hipdnnSetTensorNdDescriptor(cxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	cudnnErrCheck(hipdnnSetTensorNdDescriptor(hyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	cudnnErrCheck(hipdnnSetTensorNdDescriptor(cyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
	
	// -------------------------
	// Set up the dropout descriptor (needed for the RNN descriptor)
	// -------------------------
	unsigned long long seed = 1337ull; // Pick a seed.

	hipdnnDropoutDescriptor_t dropoutDesc;
	cudnnErrCheck(hipdnnCreateDropoutDescriptor(&dropoutDesc));

	// How much memory does dropout need for states?
	// These states are used to generate random numbers internally
	// and should not be freed until the RNN descriptor is no longer used
	size_t stateSize;
	void *states;
	cudnnErrCheck(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));
	cudaErrCheck(hipMalloc(&states, stateSize));
	cudnnErrCheck(hipdnnSetDropoutDescriptor(dropoutDesc, cudnnHandle, dropout, states, stateSize, seed));

	// -------------------------   
	// Set up the RNN descriptor
	// -------------------------
	hipdnnRNNDescriptor_t rnnDesc;
	hipdnnRNNMode_t RNNMode;

	cudnnErrCheck(hipdnnCreateRNNDescriptor(&rnnDesc));

	if (mode == 0) RNNMode = HIPDNN_RNN_RELU;
	else if (mode == 1) RNNMode = HIPDNN_RNN_TANH;
	else if (mode == 2) RNNMode = HIPDNN_LSTM;
	else if (mode == 3) RNNMode = HIPDNN_GRU;

	cudnnErrCheck(hipdnnSetRNNDescriptor(cudnnHandle, rnnDesc,
		hiddenSize,
		numLayers,
		dropoutDesc,
		HIPDNN_LINEAR_INPUT, // We can also skip the input matrix transformation
		bidirectional ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL,
		RNNMode, HIPDNN_RNN_ALGO_STANDARD,
		HIPDNN_DATA_FLOAT));

	// -------------------------
	// Set up parameters
	// -------------------------
	// This needs to be done after the rnn descriptor is set as otherwise
	// we don't know how many parameters we have to allocate
	void *w;
	void *dw;

	hipdnnFilterDescriptor_t wDesc, dwDesc;

	cudnnErrCheck(hipdnnCreateFilterDescriptor(&wDesc));
	cudnnErrCheck(hipdnnCreateFilterDescriptor(&dwDesc));

	size_t weightsSize;
	cudnnErrCheck(hipdnnGetRNNParamsSize(cudnnHandle, rnnDesc, xDesc[0], &weightsSize, HIPDNN_DATA_FLOAT));

	int dimW[3];
	dimW[0] = (int)floor(weightsSize / sizeof(float));
	dimW[1] = 1;
	dimW[2] = 1;

	cudnnErrCheck(hipdnnSetFilterNdDescriptor(wDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, dimW));
	cudnnErrCheck(hipdnnSetFilterNdDescriptor(dwDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, dimW));

	cudaErrCheck(hipMalloc((void**)&w, weightsSize));
	cudaErrCheck(hipMalloc((void**)&dw, weightsSize));


	// -------------------------
	// Set up work space and reserved memory
	// -------------------------   
	void *workspace;
	void *reserveSpace;

	size_t workSize;
	size_t reserveSize;

	// Need for every pass
	cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc, seqLength, xDesc, &workSize));
	// Only needed in training, shouldn't be touched between passes.
	cudnnErrCheck(hipdnnGetRNNTrainingReserveSize(cudnnHandle, rnnDesc, seqLength, xDesc, &reserveSize));

	cudaErrCheck(hipMalloc((void**)&workspace, workSize));
	cudaErrCheck(hipMalloc((void**)&reserveSpace, reserveSize));

	// Weights
	int numLinearLayers = 0;
	if (RNNMode == HIPDNN_RNN_RELU || RNNMode == HIPDNN_RNN_TANH) {
		numLinearLayers = 2;
	}
	else if (RNNMode == HIPDNN_LSTM) {
		numLinearLayers = 8;
	}
	else if (RNNMode == HIPDNN_GRU) {
		numLinearLayers = 6;
	}

	for (int layer = 0; layer < numLayers * (bidirectional ? 2 : 1); layer++) {
		for (int linLayerID = 0; linLayerID < numLinearLayers; linLayerID++) {
			hipdnnFilterDescriptor_t linLayerMatDesc;
			cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerMatDesc));
			float *linLayerMat;

			cudnnErrCheck(hipdnnGetRNNLinLayerMatrixParams(cudnnHandle, rnnDesc, layer, xDesc[0], wDesc, w, linLayerID, linLayerMatDesc, (void**)&linLayerMat));

			hipdnnDataType_t dataType;
			hipdnnTensorFormat_t format;
			int nbDims;
			int filterDimA[3];
			cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerMatDesc, 3, &dataType, &format, &nbDims, filterDimA));

			initGPUData(linLayerMat, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f / (float)(filterDimA[0] * filterDimA[1] * filterDimA[2]));

			cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerMatDesc));

			hipdnnFilterDescriptor_t linLayerBiasDesc;
			cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerBiasDesc));
			float *linLayerBias;

			cudnnErrCheck(hipdnnGetRNNLinLayerBiasParams(cudnnHandle, rnnDesc, layer, xDesc[0], wDesc, w, linLayerID, linLayerBiasDesc, (void**)&linLayerBias));

			cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerBiasDesc, 3, &dataType, &format, &nbDims, filterDimA));

			initGPUData(linLayerBias, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f);

			cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerBiasDesc));
		}
	}

	// *********************************************************************************************************
	// At this point all of the setup is done. We now need to pass through the RNN (one batch at a time).
	// *********************************************************************************************************
	for (int b = 0; b < batchCount; b++) {
		//-- Load input and output (target) data
		cudaErrCheck(hipMemcpy(x, hSample[b], (batchSize*seqLength*inputSize) * sizeof(float), hipMemcpyHostToDevice));
		cudaErrCheck(hipMemcpy(y, hTarget[b], (batchSize*seqLength*inputSize) * sizeof(float), hipMemcpyHostToDevice));
		//-- FeedForward
		cudnnErrCheck(hipdnnRNNForwardTraining(cudnnHandle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSize, reserveSpace, reserveSize));
		//-- Feedback Context
		cudnnErrCheck(hipdnnRNNBackwardData(cudnnHandle, rnnDesc, seqLength, yDesc, y, dyDesc, dy, dhyDesc, dhy, dcyDesc, dcy, wDesc, w, hxDesc, hx, cxDesc, cx, dxDesc, dx, dhxDesc, dhx, dcxDesc, dcx, workspace, workSize, reserveSpace, reserveSize));
		//-- calc and apply dW
		cudaErrCheck(hipMemset(dw, 0, weightsSize));
		cudnnErrCheck(hipdnnRNNBackwardWeights(cudnnHandle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, yDesc, y, workspace, workSize, dwDesc, dw, reserveSpace, reserveSize));
		//-- FF to test data
		cudnnErrCheck(hipdnnRNNForwardInference(cudnnHandle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSize));
		//-- sync (?)
		cudaErrCheck(hipDeviceSynchronize());
	}

	return 0;
}
EXPORT int RNNtrain(int sqLen, int inputSize, int hiddenSize, int numLayers, int batchCount, int batchSize, bool bidirectional, int mode, float dropout, float** sample, float** target) {
	
	setupGPU(
		sqLen, inputSize, batchCount, batchSize, numLayers, hiddenSize, bidirectional, mode, dropout, sample, target
	);
	//===

	//=== 2. Load Input Data (one batch at a time)

	//-- sample and target are both in OHLC/OHLCV format (inputSize=4/5)
	// sample[batchCount][batchSize][seqLen][inputSize]
	// target[batchCount][batchSize][seqLen][inputSize]

	return 0;
}

int RNNrun() {
	return 0;
}